#include "hip/hip_runtime.h"
// This code is written at BigVision LLC. It is based on the OpenCV project.
//It is subject to the license terms in the LICENSE file found in this distribution and at http://opencv.org/license.html

// Usage example:  ./object_detection_yolo.out --video=run.mp4
//                 ./object_detection_yolo.out --image=bird.jpg
#include <fstream>
#include <sstream>
#include <iostream>

#include <opencv2/dnn.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
;
using namespace cv;
using namespace dnn;
using namespace std;

string pro_dir = "D:/darknet-master/build/darknet/x64/"; //��Ŀ��Ŀ¼
String modelConfigurationDefault = pro_dir + "cfg/yolov3.cfg";
String modelWeightsDefault = pro_dir + "yolov3.weights";
string image_path = "D:/text/02.jpg";
string classesFileDefault = pro_dir + "data/coco.names";// "coco.names";
string video_path = "D:/Gymshark.avi";

// Initialize the parameters
float confThreshold = 0.5; // Confidence threshold
float nmsThreshold = 0.4;  // Non-maximum suppression threshold
int inpWidth = 416;  // Width of network's input image
int inpHeight = 416; // Height of network's input image
vector<string> classes;

//netӦ�����ȫ�ֱ���
Net net;

// Remove the bounding boxes with low confidence using non-maxima suppression
void postprocess(Mat& frame, const vector<Mat>& outs, vector<int>& indices, vector<int>& classIds, vector<float>& confidences, vector<Rect>& boxes);
void postprocess(Mat& frame, const vector<Mat>& outs);
// Draw the predicted bounding box
void drawPred(int classId, float conf, int left, int top, int right, int bottom, Mat& frame);

// Get the names of the output layers
vector<String> getOutputsNames(const Net& net);

void detect_image(string image_path, string modelWeights, string modelConfiguration, string classesFile);

void detect_video(string video_path, string modelWeights, string modelConfiguration, string classesFile);

vector<string> loadModel(string modelWeights = modelWeightsDefault, string modelConfiguration = modelConfigurationDefault, string classesFile = classesFileDefault);
void objectDetection(Mat leftImg, vector<int>& indices, vector<int>& classIds, vector<Rect>& boxes);

void detect_image(string image_path, string modelWeights, string modelConfiguration, string classesFile) {
	// Load names of classes
	ifstream ifs(classesFile.c_str());
	string line;
	while (getline(ifs, line)) classes.push_back(line);

	// Load the network
	Net net = readNetFromDarknet(modelConfiguration, modelWeights);
	net.setPreferableBackend(DNN_BACKEND_CUDA);
	net.setPreferableTarget(DNN_TARGET_CUDA);

	// Open a video file or an image file or a camera stream.
	string str, outputFile;
	cv::Mat frame = cv::imread(image_path);
	// Create a window
	static const string kWinName = "Deep learning object detection in OpenCV";
	namedWindow(kWinName, WINDOW_NORMAL);

	// Stop the program if reached end of video
	// Create a 4D blob from a frame.
	Mat blob;
	blobFromImage(frame, blob, 1 / 255.0, Size(inpWidth, inpHeight), Scalar(0, 0, 0), true, false);

	//Sets the input to the network
	net.setInput(blob);

	// Runs the forward pass to get output of the output layers
	vector<Mat> outs;
	net.forward(outs, getOutputsNames(net));

	// Remove the bounding boxes with low confidence
	postprocess(frame, outs);
	// Put efficiency information. The function getPerfProfile returns the overall time for inference(t) and the timings for each of the layers(in layersTimes)
	vector<double> layersTimes;
	double freq = getTickFrequency() / 1000;
	double t = net.getPerfProfile(layersTimes) / freq;
	string label = format("Inference time for a frame : %.2f ms", t);
	putText(frame, label, Point(0, 15), FONT_HERSHEY_SIMPLEX, 0.5, Scalar(0, 0, 255));
	// Write the frame with the detection boxes
	imshow(kWinName, frame);
	cv::waitKey(30);
}

void detect_video(string video_path, string modelWeights, string modelConfiguration, string classesFile) {
	string outputFile = "output.avi";;
	// Load names of classes
	ifstream ifs(classesFile.c_str());
	string line;
	while (getline(ifs, line)) classes.push_back(line);

	// Load the network
	Net net = readNetFromDarknet(modelConfiguration, modelWeights);
	net.setPreferableBackend(DNN_BACKEND_CUDA);
	net.setPreferableTarget(DNN_TARGET_CUDA);


	// Open a video file or an image file or a camera stream.
	VideoCapture cap;
	//VideoWriter video;
	Mat frame, blob;

	try {
		// Open the video file
		ifstream ifile(video_path);
		if (!ifile) throw("error");
		cap.open(video_path);
		if (cap.isOpened())
		{
			cout << "success open mp4" << std::endl;
		}
	}
	catch (...) {
		cout << "Could not open the input image/video stream" << endl;
		return;
	}

	// Get the video writer initialized to save the output video
	//video.open(outputFile, 
	//	VideoWriter::fourcc('M', 'J', 'P', 'G'), 
	//	28, 
	//	Size(cap.get(CAP_PROP_FRAME_WIDTH), cap.get(CAP_PROP_FRAME_HEIGHT)));

	// Create a window
	static const string kWinName = "Deep learning object detection in OpenCV";
	namedWindow(kWinName, WINDOW_NORMAL);

	// Process frames.
	while (waitKey(300) < 0)
	{
		// get frame from the video
		int res = cap.read(frame);

		// Stop the program if reached end of video
		if (frame.empty()) {
			cout << "Done processing !!!" << endl;
			cout << "Output file is stored as " << outputFile << endl;
			waitKey(300);
			break;
		}
		// Create a 4D blob from a frame.
		blobFromImage(frame, blob, 1 / 255.0, Size(inpWidth, inpHeight), Scalar(0, 0, 0), true, false);

		//Sets the input to the network
		net.setInput(blob);

		// Runs the forward pass to get output of the output layers
		vector<Mat> outs;
		net.forward(outs, getOutputsNames(net));

		// Remove the bounding boxes with low confidence
		postprocess(frame, outs);

		// Put efficiency information. The function getPerfProfile returns the overall time for inference(t) and the timings for each of the layers(in layersTimes)
		vector<double> layersTimes;
		double freq = getTickFrequency() / 1000;
		double t = net.getPerfProfile(layersTimes) / freq;
		string label = format("Inference time for a frame : %.2f ms", t);
		putText(frame, label, Point(0, 15), FONT_HERSHEY_SIMPLEX, 0.5, Scalar(0, 0, 255));

		// Write the frame with the detection boxes
		Mat detectedFrame;
		frame.convertTo(detectedFrame, CV_8U);
		//video.write(detectedFrame);
		imshow(kWinName, frame);

	}

	cap.release();
	//video.release();

}

// Remove the bounding boxes with low confidence using non-maxima suppression
void postprocess(Mat& frame, const vector<Mat>& outs,vector<int>& indices, vector<int>& classIds, vector<float>& confidences, vector<Rect>& boxes)
{
	//vector<int> classIds;
	//vector<float> confidences;
	//vector<Rect> boxes;

	for (size_t i = 0; i < outs.size(); ++i)
	{
		// Scan through all the bounding boxes output from the network and keep only the
		// ones with high confidence scores. Assign the box's class label as the class
		// with the highest score for the box.
		float* data = (float*)outs[i].data;
		for (int j = 0; j < outs[i].rows; ++j, data += outs[i].cols)
		{
			Mat scores = outs[i].row(j).colRange(5, outs[i].cols);
			Point classIdPoint;
			double confidence;
			// Get the value and location of the maximum score
			minMaxLoc(scores, 0, &confidence, 0, &classIdPoint);
			if (confidence > confThreshold)
			{
				int centerX = (int)(data[0] * frame.cols);
				int centerY = (int)(data[1] * frame.rows);
				int width = (int)(data[2] * frame.cols);
				int height = (int)(data[3] * frame.rows);
				int left = centerX - width / 2;
				int top = centerY - height / 2;

				classIds.push_back(classIdPoint.x);
				confidences.push_back((float)confidence);
				boxes.push_back(Rect(left, top, width, height));
			}
		}
	}

	// Perform non maximum suppression to eliminate redundant overlapping boxes with
	// lower confidences
	//vector<int> indices;
	NMSBoxes(boxes, confidences, confThreshold, nmsThreshold, indices);
	for (size_t i = 0; i < indices.size(); ++i)
	{
		int idx = indices[i];
		Rect box = boxes[idx];
		drawPred(classIds[idx], confidences[idx], box.x, box.y,
			box.x + box.width, box.y + box.height, frame);
	}
}

void postprocess(Mat& frame, const vector<Mat>& outs)
{
	vector<int> classIds;
	vector<float> confidences;
	vector<Rect> boxes;

	for (size_t i = 0; i < outs.size(); ++i)
	{
		// Scan through all the bounding boxes output from the network and keep only the
		// ones with high confidence scores. Assign the box's class label as the class
		// with the highest score for the box.
		float* data = (float*)outs[i].data;
		for (int j = 0; j < outs[i].rows; ++j, data += outs[i].cols)
		{
			Mat scores = outs[i].row(j).colRange(5, outs[i].cols);
			Point classIdPoint;
			double confidence;
			// Get the value and location of the maximum score
			minMaxLoc(scores, 0, &confidence, 0, &classIdPoint);
			if (confidence > confThreshold)
			{
				int centerX = (int)(data[0] * frame.cols);
				int centerY = (int)(data[1] * frame.rows);
				int width = (int)(data[2] * frame.cols);
				int height = (int)(data[3] * frame.rows);
				int left = centerX - width / 2;
				int top = centerY - height / 2;

				classIds.push_back(classIdPoint.x);
				confidences.push_back((float)confidence);
				boxes.push_back(Rect(left, top, width, height));
			}
		}
	}

	// Perform non maximum suppression to eliminate redundant overlapping boxes with
	// lower confidences
	vector<int> indices;
	NMSBoxes(boxes, confidences, confThreshold, nmsThreshold, indices);
	for (size_t i = 0; i < indices.size(); ++i)
	{
		int idx = indices[i];
		Rect box = boxes[idx];
		drawPred(classIds[idx], confidences[idx], box.x, box.y,
			box.x + box.width, box.y + box.height, frame);
	}
}
// Draw the predicted bounding box
void drawPred(int classId, float conf, int left, int top, int right, int bottom, Mat& frame)
{
	//Draw a rectangle displaying the bounding box
	rectangle(frame, Point(left, top), Point(right, bottom), Scalar(255, 178, 50), 3);

	//Get the label for the class name and its confidence
	string label = format("%.2f", conf);
	if (!classes.empty())
	{
		CV_Assert(classId < (int)classes.size());
		label = classes[classId] + ":" + label;
	}

	//Display the label at the top of the bounding box
	int baseLine;
	Size labelSize = getTextSize(label, FONT_HERSHEY_SIMPLEX, 0.5, 1, &baseLine);
	top = max(top, labelSize.height);
	rectangle(frame, Point(left, top - round(1.5*labelSize.height)), Point(left + round(1.5*labelSize.width), top + baseLine), Scalar(255, 255, 255), FILLED);
	//putText(frame, label, Point(left, top), FONT_HERSHEY_SIMPLEX, 0.75, Scalar(0, 0, 0), 1);
}

// Get the names of the output layers
vector<String> getOutputsNames(const Net& net)
{
	static vector<String> names;
	if (names.empty())
	{
		//Get the indices of the output layers, i.e. the layers with unconnected outputs
		vector<int> outLayers = net.getUnconnectedOutLayers();

		//get the names of all the layers in the network
		vector<String> layersNames = net.getLayerNames();

		// Get the names of the output layers in names
		names.resize(outLayers.size());
		for (size_t i = 0; i < outLayers.size(); ++i)
			names[i] = layersNames[outLayers[i] - 1];
	}
	return names;
}

vector<string> loadModel(string modelWeights, string modelConfiguration, string classesFile)
{
	ifstream ifs(classesFile.c_str());
	string line;
	while (getline(ifs, line)) classes.push_back(line);

	// Load the network
	//netӦ�����ȫ�ֱ���
	net = readNetFromDarknet(modelConfiguration, modelWeights);
	net.setPreferableBackend(DNN_BACKEND_CUDA);
	net.setPreferableTarget(DNN_TARGET_CUDA);
	return classes;
}
void objectDetection(Mat leftImg, vector<int>& indices, vector<int>& classIds,vector<Rect>& boxes)
{
	Mat ouputleftImg; 
	leftImg.copyTo(ouputleftImg);
	static const string kWinName = "Deep learning object detection in OpenCV";
	namedWindow(kWinName, WINDOW_NORMAL);

	// Stop the program if reached end of video
	// Create a 4D blob from a frame.
	Mat blob;
	blobFromImage(ouputleftImg, blob, 1 / 255.0, Size(inpWidth, inpHeight), Scalar(0, 0, 0), true, false);

	//Sets the input to the network
	net.setInput(blob);

	// Runs the forward pass to get output of the output layers
	vector<Mat> outs;
	//vector<int> indices;
	//vector<int> classIds;
	vector<float> confidences;
	//vector<Rect> boxes;
	net.forward(outs, getOutputsNames(net));

	// Remove the bounding boxes with low confidence
	postprocess(ouputleftImg, outs, indices, classIds, confidences, boxes);
	//ѭ��������ֵ(����)
	/*
	for (size_t i = 0; i < indices.size(); ++i)
	{
		int idx = indices[i];
		Rect box = boxes[idx];
		float confident = confidences[idx];
		int classid = classIds[idx];
		string confient_string = format("%.2f", confident);
		cout << classes[classid] << "  " << confident << " "<< box << endl;
		
	}
	*/
	// Put efficiency information. The function getPerfProfile returns the overall time for inference(t) and the timings for each of the layers(in layersTimes)
	vector<double> layersTimes;
	double freq = getTickFrequency() / 1000;
	double t = net.getPerfProfile(layersTimes) / freq;
	string label = format("Inference time for a frame : %.2f ms", t);
	putText(ouputleftImg, label, Point(0, 15), FONT_HERSHEY_SIMPLEX, 0.5, Scalar(0, 0, 255));
	// Write the frame with the detection boxes
	imshow(kWinName, ouputleftImg);
	cv::waitKey(30);
}